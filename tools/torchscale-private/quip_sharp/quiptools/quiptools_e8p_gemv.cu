#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <vector>
#include <utility>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace torch::indexing;
using namespace nvcuda;

#define FULL_MASK 0xffffffff
#define HALF_MASK 0x0000ffff

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}

#define BLOCK_SIZE 512
#define WARP_SIZE 32


__device__ static inline uint64_t decode8weights(
    uint16_t weight_compressed,
    const int64_t *__restrict__ codebook_abs
) {

    uint32_t bit_shift = (weight_compressed & 1)^1;
    uint8_t bits_sign = (weight_compressed >> 1) & ((1 << 7) - 1);
    uint8_t bits_abs = (weight_compressed >> 8) & ((1 << 9) - 1);

    int64_t packed_ = codebook_abs[bits_abs];
    uint32_t packed[2];
    memcpy(packed, &packed_, sizeof(packed));

    // TODO: optimize this by redefining the bit pattern
    uint32_t parity = __popc(packed[0] & 0x04040404) ^ __popc(packed[1]&0x04040404);
    uint8_t sign_vec = bits_sign | ((__popc(bits_sign) ^ parity) << 7);
    uint32_t decoded_sign[2];
    decoded_sign[0] = sign_vec * 0x08040201ll;
    decoded_sign[1] = sign_vec * 0x80402010ll;
    decoded_sign[0] &= 0x80808080;
    decoded_sign[1] &= 0x80808080;
    decoded_sign[0] >>= 7;
    decoded_sign[1] >>= 7;
    decoded_sign[0] *= 255 - 3;
    decoded_sign[1] *= 255 - 3;
    packed[0] ^= decoded_sign[0];
    packed[1] ^= decoded_sign[1];
    packed[0] |= 0x01010101;
    packed[1] |= 0x01010101;
    packed[0] -= bit_shift * 0x02020202;
    packed[1] -= bit_shift * 0x02020202;

    memcpy(&packed_, packed, sizeof(packed));

    return packed_;
}


/*
llama 2 70B:
M N K
1 8192 8192
1 57344 8192
1 8192 28672
1 10240 8192
*/
template <typename scalar_t>
__global__ static void
__launch_bounds__(BLOCK_SIZE)
decode_matmul_e8p_kernel(
    scalar_t *__restrict__ output,
    const scalar_t *__restrict__ x,
    const int16_t *__restrict__ weights_compressed,
    const int64_t *__restrict__ codebook_abs,
    int64_t M,
    int64_t N,
    int64_t K
) {
    __shared__ int64_t codebook_local[256];
    if (threadIdx.x < 256) {
    codebook_local[threadIdx.x] = codebook_abs[threadIdx.x];
    }
    __syncthreads();

    int64_t warpId = threadIdx.x / WARP_SIZE;
    int64_t laneId = threadIdx.x % WARP_SIZE;

    // each thread adds 8 activation-weight products
    const int64_t unroll_k = 2;
    const int64_t pack = 8;
    const int64_t elem_per_thread = pack * unroll_k;
    int64_t warps_per_elem = K / WARP_SIZE / elem_per_thread;
    const int64_t unroll_n = 16;
    const int64_t local_k = 1; // in terms of warp size. 32 threads of elem_per_thread fma each, dont set below 1 because of __shfl_down_sync
    int64_t local_n = BLOCK_SIZE / WARP_SIZE / local_k;
    int64_t grid_N = N / unroll_n;

    __shared__ scalar_t accum_scratch[BLOCK_SIZE / WARP_SIZE];
    bool SHARED_REDUCE = false;

    for (int64_t warpPos = blockIdx.x * BLOCK_SIZE/WARP_SIZE + warpId;
            warpPos < M * grid_N * warps_per_elem;
            warpPos += gridDim.x * BLOCK_SIZE/WARP_SIZE) {

        int64_t local_n_i = (warpPos% (BLOCK_SIZE / WARP_SIZE)) / local_k;
        int64_t local_k_i = (warpPos% (BLOCK_SIZE / WARP_SIZE)) % local_k;
        int64_t m = (warpPos / warps_per_elem) / (grid_N);
        int64_t k_ = warpPos % (warps_per_elem * local_n);
        int64_t k = k_ / (local_k * local_n) * local_k + k_ % local_k;

        scalar_t this_activations[elem_per_thread];
#pragma unroll
        for (int64_t unroll_k_i = 0; unroll_k_i < unroll_k; unroll_k_i++) {
            const scalar_t *activations = x + m * K + (k * WARP_SIZE + laneId) * elem_per_thread + unroll_k_i * pack;
            if constexpr (std::is_same<scalar_t, float>::value) {
                const float4 *first_half = reinterpret_cast<const float4 *>(activations);
                __builtin_assume_aligned(first_half, 16);
                this_activations[unroll_k_i * pack + 0] = first_half->x;
                this_activations[unroll_k_i * pack + 1] = first_half->y;
                this_activations[unroll_k_i * pack + 2] = first_half->z;
                this_activations[unroll_k_i * pack + 3] = first_half->w;
                const float4 *second_half = reinterpret_cast<const float4 *>(activations + 4);
                __builtin_assume_aligned(second_half, 16);
                this_activations[unroll_k_i * pack + 4] = second_half->x;
                this_activations[unroll_k_i * pack + 5] = second_half->y;
                this_activations[unroll_k_i * pack + 6] = second_half->z;
                this_activations[unroll_k_i * pack + 7] = second_half->w;
            } else {
                for (int64_t activation_i = 0; activation_i < pack; activation_i++) {
                    this_activations[unroll_k_i * pack + activation_i] = activations[activation_i];
                }
            }
        }
        for (int64_t unroll_n_i = 0; unroll_n_i < unroll_n; unroll_n_i++) {
            scalar_t accumulator = 0;
            int64_t n = ((warpPos/local_k) % local_n) + ((warpPos / warps_per_elem) % grid_N) / local_n * local_n;
            __syncwarp();
            uint16_t this_weights[unroll_k];
            if (unroll_k % 2 == 0) {
                for (int64_t unroll_k_i = 0; unroll_k_i < unroll_k; unroll_k_i+=2) {
                    const ushort2 *loaded = (const ushort2 *) &weights_compressed[(n*unroll_n + unroll_n_i) * K/pack + (k * WARP_SIZE + laneId) * unroll_k + unroll_k_i];
                    __builtin_assume_aligned(loaded, 4);
                    this_weights[unroll_k_i] = loaded->x;
                    this_weights[unroll_k_i + 1] = loaded->y;
                }
            } else {
                for (int64_t unroll_k_i = 0; unroll_k_i < unroll_k; unroll_k_i++) {
                    this_weights[unroll_k_i] = weights_compressed[(n*unroll_n + unroll_n_i) * K/pack + (k * WARP_SIZE + laneId) * unroll_k + unroll_k_i];
                }
            }

#pragma unroll
            for (int64_t unroll_k_i = 0; unroll_k_i < unroll_k; unroll_k_i++) {
                // TODO: optimize access pattern by reordering weights
                uint16_t encoded = this_weights[unroll_k_i];
                uint64_t decoded = decode8weights(encoded, codebook_local);

                #ifdef EMULATED_INT82FP16
                // bit twiddling to convert int8 to fp16 from http://arxiv.org/abs/2211.10017
                half2 unpacked[2][2];
                uint64_t lower_half = decoded & 0x00ff00ff00ff00ff;
                lower_half = (lower_half ^ 0x6480648064806480);
                memcpy(unpacked[0], &lower_half, sizeof(uint64_t));
                uint64_t upper_half = (decoded & 0xff00ff00ff00ff00) >> 8;
                upper_half = (upper_half ^ 0x6480648064806480);
                memcpy(unpacked[1], &upper_half, sizeof(uint64_t));

                const half2 adjust = {__float2half(-1152.0f), __float2half(-1152.0f)};
                unpacked[0][0] = __hadd2(unpacked[0][0], adjust);
                unpacked[0][1] = __hadd2(unpacked[0][1], adjust);
                unpacked[1][0] = __hadd2(unpacked[1][0], adjust);
                unpacked[1][1] = __hadd2(unpacked[1][1], adjust);

                float2 unpacked_f[2][2];
                unpacked_f[0][0] = __half22float2(unpacked[0][0]);
                unpacked_f[0][1] = __half22float2(unpacked[0][1]);
                unpacked_f[1][0] = __half22float2(unpacked[1][0]);
                unpacked_f[1][1] = __half22float2(unpacked[1][1]);


                accumulator += this_activations[unroll_k_i * pack + 0] * (unpacked_f[0][0].x);
                accumulator += this_activations[unroll_k_i * pack + 1] * (unpacked_f[1][0].x);
                accumulator += this_activations[unroll_k_i * pack + 2] * (unpacked_f[0][0].y);
                accumulator += this_activations[unroll_k_i * pack + 3] * (unpacked_f[1][0].y);
                accumulator += this_activations[unroll_k_i * pack + 4] * (unpacked_f[0][1].x);
                accumulator += this_activations[unroll_k_i * pack + 5] * (unpacked_f[1][1].x);
                accumulator += this_activations[unroll_k_i * pack + 6] * (unpacked_f[0][1].y);
                accumulator += this_activations[unroll_k_i * pack + 7] * (unpacked_f[1][1].y);
                #else
                for (int64_t i = 0; i < 8; i += 1) {
                    int8_t weight = decoded >> (i * 8);
                    accumulator += this_activations[unroll_k_i * pack + i] * (int8_t) weight;
                }
                #endif
            }
            accumulator *= 0.25;

            for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
                // apparently c10::Half does arithmetic operations in float32?
                // https://github.com/pytorch/pytorch/blob/0bd4d1f4ab38d3088de8aa5fbba35427b42d118e/c10/util/Half.h#L4C58-L6C80
                if constexpr (std::is_same<scalar_t, c10::Half>::value) {
                    accumulator += __shfl_down_sync(0xFFFFFFFF, __float2half(accumulator), offset);
                } else {
                    accumulator += __shfl_down_sync(0xFFFFFFFF, accumulator, offset);
                }
            }

            if (SHARED_REDUCE) {
                if (laneId == 0) {
                    accum_scratch[warpId] = accumulator;
                    __syncthreads();
                    if (warpId % local_k == 0) {
                        scalar_t local_accum = 0;
                        for (int64_t accum_i = 0; accum_i < local_k; accum_i++) {
                            local_accum += accum_scratch[warpId / local_k * local_k + accum_i];
                        }
                        atomicAdd(output + m * N + n * unroll_n + unroll_n_i, local_accum);
                    }
                } else {
                    __syncthreads();
                }
            } else {
                if (laneId == 0) {
                    atomicAdd(output + m * N + n * unroll_n + unroll_n_i, accumulator);
                }
            }
        }
    }
}


__host__ extern torch::Tensor decode_matmul_e8p(
    torch::Tensor x,
    torch::Tensor weights_compressed,
    torch::Tensor codebook_abs
) {

    CHECK_INPUT(x);
    CHECK_INPUT(weights_compressed);
    CHECK_INPUT(codebook_abs);

    TORCH_CHECK(weights_compressed.scalar_type() == torch::kInt16);
    TORCH_CHECK(codebook_abs.scalar_type() == torch::kInt64);
    TORCH_CHECK(x.size(-1) == weights_compressed.size(-1) << 3);
    TORCH_CHECK(codebook_abs.size(-1) == 256);

    int64_t M = x.size(-2);
    int64_t N = weights_compressed.size(-2);
    int64_t K = x.size(-1);
    //printf("%lld %lld %lld\n", M, N, K);

    TORCH_CHECK(K % WARP_SIZE == 0, "K is not divisible by WARP_SIZE");

    at::DeviceGuard guard(x.device());
    torch::TensorOptions options = torch::TensorOptions()
        .dtype(x.scalar_type())
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{M, N}, options);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, x.get_device());
    int64_t grid_size = static_cast<int64_t>(6 * deviceProp.multiProcessorCount);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND2(
            at::ScalarType::Half,
            at::ScalarType::BFloat16,
            x.scalar_type(),
            "decode_matmul_e8p",
            [&] {
        decode_matmul_e8p_kernel<<<grid_size, BLOCK_SIZE, 0, stream>>>(
                output.data_ptr<scalar_t>(),
                x.data_ptr<scalar_t>(),
                weights_compressed.data_ptr<int16_t>(),
                codebook_abs.data_ptr<int64_t>(),
                M,
                N,
                K);
        gpuErrchk(hipPeekAtLastError());
    });

    return output;
}
